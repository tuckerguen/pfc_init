#include "hip/hip_runtime.h"
//
// Created by tucker on 5/19/20.
//
#include <opencv2/highgui.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <iostream>
#include <stdio.h>
#include "NeedleMatch.hpp"

using namespace std;
using namespace cv;

string image_window = "Source Image";
string result_window = "Result Window";
string template_window = "Template";

//A template match
struct match {
    double maxVal; //max matching value assigned by opencv templatematch()
    double angle; //angle matched at
    double scale; //scale matched at
    Point maxLoc; //location of match
    Mat templ; //template used to match
};

int match_method;
int max_Trackbar = 5;

int main(int argc, char** argv){
    namedWindow( image_window, WINDOW_AUTOSIZE );
    namedWindow( result_window, WINDOW_AUTOSIZE );
    namedWindow( template_window,  WINDOW_AUTOSIZE );

    Mat raw_img, raw_templ, img_hsv, templ_hsv;
    Mat *img, *templ, result, *iTempl;

    // match match bestMatch = {
    //     -DBL_MAX,
    //     0.0,
    //     0.0,
    //     Point(0, 0),
    //     templ,
    // };
    
    size_t img_size = img.total() * img.elemSize();
    size_t templ_size = templ.total() * templ.elemSize();
    size_t iTempl_size = iTempl.total() * iTempl.elemSize();
    //Add necessary mats to unified memory
    hipMallocManaged(&img, img_size);
    hipMallocManaged(&templ, templ_size);
    hipMallocManaged(&iTempl, iTempl_size);

    //pick block size
    int blockSize = 256;
    int numBlocks = (360 + blockSize - 1) / blockSize;
    
    //CPU image preproc operations
    raw_img = imread("../../imgs/raw_l_c.png", IMREAD_COLOR);
    raw_templ = imread("../../imgs/raw_l_b.png", IMREAD_COLOR);
    Rect r(168, 92, 58, 35);
    raw_templ = raw_templ(r);
    cvtColor(raw_img, img_hsv, COLOR_BGR2HSV);
    inRange(img_hsv, Scalar(0, 0, 0), Scalar(5, 0, 140), img);
    cvtColor(raw_templ, templ_hsv, COLOR_BGR2HSV);
    inRange(templ_hsv, Scalar(0, 0, 0), Scalar(5, 0, 140), iTempl);

    //Run Rotations and matching on the GPU
    run<<<numBlocks, blockSize>>>();

    imshow(template_window, templ);

    waitKey();

    return 0;
}

__global__ run(template){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < 360; i+= stride){
        RotTemplate(i);
        MatchImageToTemplate();
    }
}

__device__ void RotTemplate(double angle){
    // get rotation matrix for rotating the image around its center in pixel coordinates
    Point2f center((iTempl.cols-1)/2.0, (iTempl.rows-1)/2.0);
    Mat rot = getRotationMatrix2D(center, angle, 1.0);
    // determine bounding rectangle, center not relevant
    Rect2f bbox = RotatedRect(Point2f(), iTempl.size(), angle).boundingRect2f();
    // adjust transformation matrix
    rot.at<double>(0,2) += bbox.width/2.0 - iTempl.cols/2.0;
    rot.at<double>(1,2) += bbox.height/2.0 - iTempl.rows/2.0;

    warpAffine(iTempl, templ, rot, bbox.size());
}

__device__ void MatchImageToTemplate(){
    Mat img_display;
    img_display = img.clone();

    /// Create the result matrix
    int result_cols =  img.cols - templ.cols + 1;
    int result_rows = img.rows - templ.rows + 1;

    result.create( result_rows, result_cols, CV_32FC1 );

    /// Do the Matching and Normalize
    matchTemplate( img, templ, result, TM_CCOEFF );
    normalize( result, result, 0, 1, NORM_MINMAX, -1, Mat() );

    /// Localizing the best match with minMaxLoc
    double minVal; double maxVal; Point minLoc; Point maxLoc;
    Point matchLoc;

    minMaxLoc( result, &minVal, &maxVal, &minLoc, &maxLoc, Mat() );

    /// For SQDIFF and SQDIFF_NORMED, the best matches are lower values. For all the other methods, the higher the better
    if( match_method  == TM_SQDIFF || match_method == TM_SQDIFF_NORMED )
    { matchLoc = minLoc; }
    else
    { matchLoc = maxLoc; }

    /// Show me what you got
    rectangle( img, matchLoc, Point( matchLoc.x + templ.cols , matchLoc.y + templ.rows ), Scalar::all(255), 2, 8, 0 );
    rectangle( result, matchLoc, Point( matchLoc.x + templ.cols , matchLoc.y + templ.rows ), Scalar::all(0), 2, 8, 0 );

    imshow( image_window, img );
    imshow( result_window, result );

    return;
}